#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include "../include/kmeans_gpu1.cuh"
#include <iostream>

#define BLOCK_SIZE 256

// Macro for checking CUDA errors
#define CUDA_CHECK(call, res) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            cleanup_gpu_resources(res); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Cleanup function
void cleanup_gpu_resources(GPUResources& res) {
    if (res.d_points) hipFree(res.d_points);
    if (res.d_centroids) hipFree(res.d_centroids);
    if (res.d_new_centroids) hipFree(res.d_new_centroids);
    if (res.d_assignments) hipFree(res.d_assignments);
    if (res.d_assignments_counter) hipFree(res.d_assignments_counter);
    if (res.d_cluster_sizes) hipFree(res.d_cluster_sizes);
    if (res.d_cluster_sums) hipFree(res.d_cluster_sums);
    if (res.d_changed) hipFree(res.d_changed);
    res = GPUResources(); // Reset to nullptr
}

// CUDA kernel for calculating distances and finding nearest centroids
__global__ void findNearestCentroids(
    const float* points,      // Pointer to the array of points
    const float* centroids,   // Pointer to the array of centroids
    int* assignments,         // Pointer to the array of assignments
    const int N,              // Number of points
    const int K,              // Number of clusters
    const int D,              // Number of dimensions
    int* changed              // Number of changed assignments
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int blockIdx = threadIdx.x;
    int points_in_block = blockDim.x;

    // Put centroids and points in shared memory
    extern __shared__ float shmem[];

    float* shared_centroids = shmem;
    float* shared_points = &shmem[K * D];

    // Threads in every block with idx.x < K will each put one cluster into shared memory
    if (threadIdx.x < K) {
        for (int i = 0; i < D; i++) {
            shared_centroids[i * K + blockIdx] = centroids[i * K + blockIdx];
        }
    }

    __syncthreads();

    if (idx < N) {
        // Each thread puts it's own point into the shared memory
        for (int i = 0; i < D; i++) {
            shared_points[i * points_in_block + blockIdx] = points[i * N + idx];
        }

        float min_dist = FLT_MAX;
        int nearest_centroid = 0;

        // Iterate over each centroid
        for (int c = 0; c < K; c++) {
            float dist = 0.0f;

            // Calculate the squared Euclidean distance
            for (int d = 0; d < D; d++) {
                float diff = shared_points[d * points_in_block + blockIdx] - shared_centroids[d * K + c];
                dist += diff * diff;
            }

            // Update the nearest centroid if a closer one is found
            if (dist < min_dist) {
                min_dist = dist;
                nearest_centroid = c;
            }
        }

        // Assign the point to the nearest centroid
        if (assignments[idx] != nearest_centroid)
            atomicAdd(changed, 1);
        assignments[idx] = nearest_centroid;
    }
}

// CUDA kernel to sum up all the points for each cluster and dimension
__global__ void sum(
    const float* points,
    const int* assignments,
    int* assignments_counter,
    float* new_centroids,
    const int N,
    const int k,
    const int d
) {
    extern __shared__ float shared_mem[];
    float* shared_sums = shared_mem;
    int* shared_counts = (int*)&shared_sums[k * d];
    
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;

    // Initialize shared memory
    for (int i = tid; i < k * d; i += blockDim.x) {
        shared_sums[i] = 0.0f;
    }
    if (tid < k) {
        shared_counts[tid] = 0;
    }
    
    __syncthreads();

    // Process points
    if (gid < N) {
        int cluster = assignments[gid];
        atomicAdd(&shared_counts[cluster], 1);
        
        for (int dim = 0; dim < d; dim++) {
            float point_val = points[dim * N + gid];
            atomicAdd(&shared_sums[dim * k + cluster], point_val);
        }
    }

    __syncthreads();

    // Add shared sums to new centroids in global memory
    for (int i = tid; i < k * d; i += blockDim.x) {
        int cluster = i % k;
        int dim = i / k;
        if (shared_sums[dim * k + cluster] != 0.0f) {
            atomicAdd(&new_centroids[dim * k + cluster], 
                     shared_sums[dim * k + cluster]);
        }
    }

    // Add shared counts to assignments counter in global memory
    if (tid < k && shared_counts[tid] > 0) {
        atomicAdd(&assignments_counter[tid], shared_counts[tid]);
    }
}

// CUDA kernel to assign the new centroids divided by assignments counters to the centroids
__global__ void update(
    float* centroids,
    float* new_centroids,
    const int* assignments_counter,
    const int K,
    const int D
) {
    const int tid = threadIdx.x;
    const int total_elements = K * D;
    
    for (int idx = tid; idx < total_elements; idx += blockDim.x) {
        int cluster = idx % K;
        int dim = idx / K;
        
        if (assignments_counter[cluster] > 0) {
            centroids[dim * K + cluster] = 
                new_centroids[dim * K + cluster] / assignments_counter[cluster];
        }
    }
}

// First version of gpu K-Means algorithm
void kmeans_gpu1(
    const float* h_points,    // Pointer to the array of points on the host
    float* h_centroids,       // Pointer to the array of centroids on the host
    int* h_assignments,       // Pointer to the array of assignments on the host
    const int N,              // Number of points
    const int d,              // Number of dimensions
    const int k,              // Number of clusters
    const int max_iter        // Maximum number of iterations
)
{
    // Create CUDA events for timing
    hipEvent_t start_it, stop_it, start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start_it);
    hipEventCreate(&stop_it);
    float milliseconds = 0.0f;
    int changed = 0;

    // Allocate device memory
    GPUResources res;
    
    CUDA_CHECK(hipMalloc(&res.d_points, N * d * sizeof(float)), res);
    CUDA_CHECK(hipMalloc(&res.d_centroids, k * d * sizeof(float)), res);
    CUDA_CHECK(hipMalloc(&res.d_assignments, N * sizeof(int)), res);
    CUDA_CHECK(hipMalloc(&res.d_changed, sizeof(int)), res);
    CUDA_CHECK(hipMalloc(&res.d_assignments_counter, k * sizeof(int)), res);
    CUDA_CHECK(hipMalloc(&res.d_new_centroids, k * d * sizeof(float)), res);
    
    // Copy data to device
    hipEvent_t start_copy, stop_copy;
    hipEventCreate(&start_copy);
    hipEventCreate(&stop_copy);
    hipEventRecord(start_copy);

    CUDA_CHECK(hipMemcpy(res.d_points, h_points, N * d * sizeof(float), hipMemcpyHostToDevice), res);
    CUDA_CHECK(hipMemcpy(res.d_centroids, h_centroids, k * d * sizeof(float), hipMemcpyHostToDevice), res);
    CUDA_CHECK(hipMemcpy(res.d_changed, &changed, sizeof(int), hipMemcpyHostToDevice), res);
    
    hipEventRecord(stop_copy);
    hipEventSynchronize(stop_copy);
    hipEventElapsedTime(&milliseconds, start_copy, stop_copy);
    std::cout << "Data copying to device: " << milliseconds << " ms" << std::endl;

    // Configure kernel launch parameters
    dim3 block_size(BLOCK_SIZE);
    dim3 num_blocks_points((N + block_size.x - 1) / block_size.x);
    dim3 num_blocks_centroids((k * d + block_size.x - 1) / block_size.x);

    // Calculate shared memory size
    size_t shared_mem_size = (k * d + BLOCK_SIZE * d) * sizeof(float);
    size_t shared_mem_size_sum = (k * d) * sizeof(float) + k * sizeof(int);

    // Create events for timing
    hipEvent_t start_kernel1, stop_kernel1, start_kernel2, stop_kernel2;
    hipEventCreate(&start_kernel1);
    hipEventCreate(&stop_kernel1);
    hipEventCreate(&start_kernel2);
    hipEventCreate(&stop_kernel2);
    
    // Main loop
    hipEventRecord(start);
    for (int iter = 0; iter < max_iter; iter++) {
        hipEventRecord(start_it);

        // Find nearest centroids
        hipEventRecord(start_kernel1);
        findNearestCentroids<<<num_blocks_points, block_size, shared_mem_size>>>(
            res.d_points, res.d_centroids, res.d_assignments, N, k, d, res.d_changed);
        
        CUDA_CHECK(hipGetLastError(), res);
        CUDA_CHECK(hipDeviceSynchronize(), res);

        hipEventRecord(stop_kernel1);
        hipEventSynchronize(stop_kernel1);
        hipEventElapsedTime(&milliseconds, start_kernel1, stop_kernel1);
        std::cout << "Assigning nearest centroids execution time: " << milliseconds << " ms" << std::endl;

        // Copy the number of changed assignments back to host
        CUDA_CHECK(hipMemcpy(&changed, res.d_changed, sizeof(int), hipMemcpyDeviceToHost), res);

        // Check if any assignments changed
        if (changed == 0) {
            std::cout << "No changes in assignments, stopping the algorithm" << std::endl;
            break;
        }

        // Reset assignemnts counters and new centroids
        CUDA_CHECK(hipMemset(res.d_assignments_counter, 0, k * sizeof(int)), res);
        CUDA_CHECK(hipMemset(res.d_new_centroids, 0, k * d * sizeof(float)), res);
        
        // Update centroids
        hipEventRecord(start_kernel2);

        // Sum all points for all clusters and dimensions using sum kernel
        sum<<<num_blocks_points, block_size, shared_mem_size_sum>>>(
            res.d_points, res.d_assignments, res.d_assignments_counter, res.d_new_centroids, N, k, d);

        CUDA_CHECK(hipGetLastError(), res);
        CUDA_CHECK(hipDeviceSynchronize(), res);

        // Update centroids by dividing the sums by the number of points in each cluster
        update<<<1, block_size>>>(res.d_centroids, res.d_new_centroids, res.d_assignments_counter, k, d);

        CUDA_CHECK(hipGetLastError(), res);
        CUDA_CHECK(hipDeviceSynchronize(), res);

        hipEventRecord(stop_kernel2);

        hipEventSynchronize(stop_kernel2);
        hipEventElapsedTime(&milliseconds, start_kernel2, stop_kernel2);
        std::cout << "Updating centroids execution time: " << milliseconds << " ms" << std::endl;
    
        // Display general info about iteration
        hipEventRecord(stop_it);
        hipEventSynchronize(stop_it);
        hipEventElapsedTime(&milliseconds, start_it, stop_it);
        std::cout << "Iteration " << iter << " completed in " << milliseconds << " ms" << std::endl;
        std::cout << "Points that changed cluster: " << changed << std::endl;

        // Setting changed back to 0
        changed = 0;
        CUDA_CHECK(hipMemcpy(res.d_changed, &changed, sizeof(int), hipMemcpyHostToDevice), res);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Total execution time of the main loop: " << milliseconds << " ms" << std::endl;
    
    // Copy results back to host
    hipEvent_t start_copy_back, stop_copy_back;
    hipEventCreate(&start_copy_back);
    hipEventCreate(&stop_copy_back);
    hipEventRecord(start_copy_back);

    CUDA_CHECK(hipMemcpy(h_centroids, res.d_centroids, k * d * sizeof(float), hipMemcpyDeviceToHost), res);
    CUDA_CHECK(hipMemcpy(h_assignments, res.d_assignments, N * sizeof(int), hipMemcpyDeviceToHost), res);

    hipEventRecord(stop_copy_back);
    hipEventSynchronize(stop_copy_back);
    hipEventElapsedTime(&milliseconds, start_copy_back, stop_copy_back);
    std::cout << "Data copying back to host: " << milliseconds << " ms" << std::endl;
    
    // Cleanup
    cleanup_gpu_resources(res);
}