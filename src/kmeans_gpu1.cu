#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include "../include/kmeans_gpu1.cuh"
#include <iostream>

// TODO: add timing information
// TODO: fix the makefile

// Macro for checking CUDA errors
#define CUDA_CHECK(call, res) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err)); \
            cleanup_gpu_resources(res); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Cleanup function
static void cleanup_gpu_resources(GPUResources& res) {
    if (res.d_points) hipFree(res.d_points);
    if (res.d_centroids) hipFree(res.d_centroids);
    if (res.d_assignments) hipFree(res.d_assignments);
    if (res.d_cluster_sizes) hipFree(res.d_cluster_sizes);
    res = GPUResources(); // Reset to nullptr
}

// CUDA kernel for calculating distances and finding nearest centroids
__global__ void findNearestCentroids(
    const float* points,      // Pointer to the array of points
    const float* centroids,   // Pointer to the array of centroids
    int* assignments,         // Pointer to the array of assignments
    const int n_points,       // Number of points
    const int n_clusters,     // Number of clusters
    const int n_dims,         // Number of dimensions
    int* changed              // Number of changed assignments
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int sidx = threadIdx.x;
    int points_in_block = blockDim.x;

    // Put centroids and points in shared memory
    __shared__ float shared_centroids[n_clusters * n_dims];
    __shared__ float shared_points[points_in_block * n_dims];

    // Threads in every block with idx.x < n_clusters will each put one cluster into shared memory
    if (threadIdx.x < n_clusters) {
        for (int i = 0; i < n_dims; i++) {
            shared_centroids[i * n_clusters + threadIdx.x] = centroids[i * n_clusters + threadIdx.x];
        }
    }

    __syncthreads();

    if (idx < n_points) {
        // Each thread puts it's own point into the shared memory
        for (int i = 0; i < n_dims; i++) {
            shared_points[i * points_in_block + sidx] = points[i * n_points + idx];
        }

        float min_dist = FLT_MAX;
        int nearest_centroid = 0;

        // Iterate over each centroid
        for (int c = 0; c < n_clusters; c++) {
            float dist = 0.0f;

            // Calculate the squared Euclidean distance
            for (int d = 0; d < n_dims; d++) {
                float diff = shared_points[d * points_in_block + sidx] - shared_centroids[d * n_clusters + c];
                dist += diff * diff;
            }

            // Update the nearest centroid if a closer one is found
            if (dist < min_dist) {
                min_dist = dist;
                nearest_centroid = c;
            }
        }

        // Assign the point to the nearest centroid
        if (assignments[idx] != nearest_centroid)
            atomicAdd(changed, 1);
        assignments[idx] = nearest_centroid;
    }
}

// CUDA kernel for updating centroids
__global__ void updateCentroids(
    const float* points,       // Pointer to the array of points
    float* centroids,          // Pointer to the array of centroids
    const int* assignments,    // Pointer to the array of assignments
    int* cluster_sizes,        // Pointer to the array of cluster sizes
    const int n_points,        // Number of points
    const int n_clusters,      // Number of clusters
    const int n_dims           // Number of dimensions
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n_clusters * n_dims) {
        int c = idx / n_dims;  // Cluster index
        int d = idx % n_dims;  // Dimension index

        // Sum up all points assigned to the cluster 'c' for dimension 'd' using sumPoints kernel
        dim3 block_size(256);
        dim3 num_blocks_points((n_points + block_size.x - 1) / block_size.x);
        sumPoints<<<num_blocks_points, block_size>>>(c, d, points, assignments, centroids, n_points, n_clusters, n_dims);
    }
}

// CUDA kernel to sum up one dimension of the points for one cluster
__global__ void sumPoints(
    const int cluster,         // Index of the cluster
    const int dimension,       // Index of the dimension
    const float* points,       // Pointer to the array of points
    const int* assignments,    // Pointer to the array of assignments
    float* centroids,          // Pointer to the array of centroids
    const int n_points,        // Number of points
    const int n_clusters,      // Number of clusters
    const int n_dims           // Number of dimensions
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Move points to shared memory (set to 0 if not assigned to the cluster)
    __shared__ float shared_points[blockDim.x];
    __shared__ int assignments_counter = 0;
    if (idx < n_points) {
        if (assignments[idx] == cluster) {
            atomicAdd(&assignments_counter, 1);
            shared_points[threadIdx.x] = points[dimension * n_points + idx];
        }
        else {
            shared_points[threadIdx.x] = 0.0f;
        }
    }

    __syncthreads();

    if (assignments_counter == 0) {
        return;
    }

    // Sum up one dimension of all the points from shared memory in parallel
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            shared_points[threadIdx.x] += shared_points[threadIdx.x + i];
            shared_points[threadIdx.x + i] = 0.0f;
        }
        if (threadIdx.x == 0 && i + i < blockDim.x) {
            shared_points[0] += shared_points[i + i];
            shared_points[i + i] = 0.0f;
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        centroids[dimension * n_clusters + cluster] = shared_points[0] / assignments_counter;
    }
}

void kmeans_gpu1(
    const float* h_points,    // Pointer to the array of points on the host
    float* h_centroids,       // Pointer to the array of centroids on the host
    int* h_assignments,       // Pointer to the array of assignments on the host
    const int N,              // Number of points
    const int d,              // Number of dimensions
    const int k,              // Number of clusters
    const int max_iter        // Maximum number of iterations
)
{
    // Create CUDA events for timing
    hipEvent_t start_it, stop_it, start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start_it);
    hipEventCreate(&stop_it);
    float milliseconds = 0.0f;

    // Allocate device memory
    GPUResources res;
    
    CUDA_CHECK(hipMalloc(&res.d_points, N * d * sizeof(float)), res);
    CUDA_CHECK(hipMalloc(&res.d_centroids, k * d * sizeof(float)), res);
    CUDA_CHECK(hipMalloc(&res.d_assignments, N * sizeof(int)), res);
    CUDA_CHECK(hipMalloc(&res.d_cluster_sizes, k * sizeof(int)), res);
    
    // Copy data to device
    hipEvent_t start_copy, stop_copy;
    hipEventCreate(&start_copy);
    hipEventCreate(&stop_copy);
    hipEventRecord(start_copy);

    CUDA_CHECK(hipMemcpy(res.d_points, h_points, N * d * sizeof(float), hipMemcpyHostToDevice), res);
    CUDA_CHECK(hipMemcpy(res.d_centroids, h_centroids, k * d * sizeof(float), hipMemcpyHostToDevice), res);
    
    hipEventRecord(stop_copy);
    hipEventSynchronize(stop_copy);
    hipEventElapsedTime(&milliseconds, start_copy, stop_copy);
    std::cout << "Data copying to device: " << milliseconds << " ms" << std::endl;

    // Configure kernel launch parameters
    dim3 block_size(256);
    dim3 num_blocks_points((N + block_size.x - 1) / block_size.x);
    dim3 num_blocks_centroids((k * d + block_size.x - 1) / block_size.x);

    // Create events for timing
    hipEvent_t start_kernel1, stop_kernel1, start_kernel2, stop_kernel2;
    hipEventCreate(&start_kernel1);
    hipEventCreate(&stop_kernel1);
    hipEventCreate(&start_kernel2);
    hipEventCreate(&stop_kernel2);
    
    // Main loop
    hipEventRecord(start);
    int changed = 0;
    for (int iter = 0; iter < max_iter; iter++) {
        hipEventRecord(start_it);

        // Find nearest centroids
        hipEventRecord(start_kernel1);
        findNearestCentroids<<<num_blocks_points, block_size>>>(
            res.d_points, res.d_centroids, res.d_assignments, N, k, d, &changed);
        hipEventRecord(stop_kernel1);
        
        // Check for kernel launch errors
        CUDA_CHECK(hipGetLastError(), res);
        
        // Wait for kernel to finish and check for errors
        CUDA_CHECK(hipDeviceSynchronize(), res);

        // Display kernel timing information
        hipEventSynchronize(stop_kernel1);
        hipEventElapsedTime(&milliseconds, start_kernel1, stop_kernel1);
        std::cout << "Assigning nearest centroids execution time: " << milliseconds << " ms" << std::endl;

        // Check if any assignments changed
        if (changed == 0) {
            std::cout << "No changes in assignments, stopping the algorithm" << std::endl;
            break;
        }
        
        // Update centroids
        hipEventRecord(start_kernel2);
        updateCentroids<<<num_blocks_centroids, block_size>>>(
            res.d_points, res.d_centroids, res.d_assignments, res.d_cluster_sizes,
            N, k, d);
        hipEventRecord(stop_kernel2);

        // Check for kernel launch errors
        CUDA_CHECK(hipGetLastError(), res);
        
        // Wait for kernel to finish and check for errors
        CUDA_CHECK(hipDeviceSynchronize(), res);

        // Display kernel timing information
        hipEventSynchronize(stop_kernel2);
        hipEventElapsedTime(&milliseconds, start_kernel2, stop_kernel2);
        std::cout << "Updating centroids execution time: " << milliseconds << " ms" << std::endl;
    
        // Display general info about iteration
        hipEventRecord(stop_it);
        hipEventSynchronize(stop_it);
        hipEventElapsedTime(&milliseconds, start_it, stop_it);
        std::cout << "Iteration " << iter << " completed in " << milliseconds << " ms" << std::endl;
        std::cout << "Points that changed cluster: " << changed << std::endl;

        changed = 0;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Total execution time of the main loop: " << milliseconds << " ms" << std::endl;
    
    // Copy results back to host
    hipEvent_t start_copy_back, stop_copy_back;
    hipEventCreate(&start_copy_back);
    hipEventCreate(&stop_copy_back);
    hipEventRecord(start_copy_back);

    CUDA_CHECK(hipMemcpy(h_centroids, res.d_centroids, k * d * sizeof(float), hipMemcpyDeviceToHost), res);
    CUDA_CHECK(hipMemcpy(h_assignments, res.d_assignments, N * sizeof(int), hipMemcpyDeviceToHost), res);

    hipEventRecord(stop_copy_back);
    hipEventSynchronize(stop_copy_back);
    hipEventElapsedTime(&milliseconds, start_copy_back, stop_copy_back);
    std::cout << "Data copying back to host: " << milliseconds << " ms" << std::endl;
    
    // Cleanup
    CUDA_CHECK(hipFree(res.d_points), res);
    CUDA_CHECK(hipFree(res.d_centroids), res);
    CUDA_CHECK(hipFree(res.d_assignments), res);
    CUDA_CHECK(hipFree(res.d_cluster_sizes), res);
}